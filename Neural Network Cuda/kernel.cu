#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "Matrix.h"
#include "NeuralNetwork.h"
#include <vector>
#include <stdio.h>

std::vector<Matrix*> Matrices;
std::vector<float*> devicePtrs;

__global__ void matrixMul(int* A, int* B, int* C,
	int aF, int aC,
	int bF, int bC,
	int cF, int cC) {
	int row = (blockIdx.y * blockDim.y) + threadIdx.y;
	int col = (blockIdx.x * blockDim.x) + threadIdx.x;
	if (aC != bF) return;
	if ((row < aF) && (col < bC)) {
		for (int k = 0; k < aC; ++k) {
			C[row * cC + col] += A[row * aC + k] * B[k * bC + col];
		}
	}
}

void prepareCuda() {
	devicePtrs.reserve(Matrices.size());
	for (size_t i = 0; i < Matrices.size(); i++)
	{
		devicePtrs.push_back(new float);
		if (hipMalloc((void**)&devicePtrs[i], sizeof(float) * Matrices[i]->columnas * Matrices[i]->filas) != hipSuccess)
			std::cout << "Error al despachar memoria a la matriz: " << i << "." << std::endl;
		// Copy data to the device
		if (hipMemcpy(devicePtrs[i], Matrices[i]->toVector(), sizeof(float) * Matrices[i]->columnas * Matrices[i]->filas, hipMemcpyHostToDevice) != hipSuccess)
			std::cout << "Error en MemCpy con la matriz: " << i << "." << std::endl;
	}
}

NeuralNetwork::NeuralNetwork(int i, int h, int o) {
	this->inputLayerNodes = i;
	this->hiddenLayerNodes = h;
	this->hiddenLayerSize = 1;
	this->outputLayerNodes = o;
	//Matriz que representa los pesos entre las capa de Entrada-Oculta
	this->pesos_ih = new Matrix(this->hiddenLayerNodes, this->inputLayerNodes);
	//Matriz que representa los pesos entre las capa Oculta-Salida
	this->pesos_ho = new Matrix(this->outputLayerNodes, this->hiddenLayerNodes);
	//Se asigna valores aleatorios a las matrices
	this->pesos_ih->aleatorizar();
	this->pesos_ho->aleatorizar();
	//Se asigna un sesgo o predisposicion a las neuronas
	this->bias_h = new Matrix(this->hiddenLayerNodes, 1);
	this->bias_o = new Matrix(this->outputLayerNodes, 1);
	//Se asigna valores aleatorios a las matrices
	this->bias_h->aleatorizar();
	this->bias_o->aleatorizar();
	this->salidas_capa_oculta = nullptr;
}

NeuralNetwork::NeuralNetwork(int i, std::vector<int>& h, int o) {
	this->inputLayerNodes = i;
	this->hiddenLayerSize = h.size();
	this->outputLayerNodes = o;
	//Matriz que representa los pesos entre las capa de Entrada-Oculta[0]
	this->pesos_ih = new Matrix(h[0], this->inputLayerNodes);
	//Se asigna valores aleatorios a las matrices
	this->pesos_ih->aleatorizar();
	//reserva espacio en memoria para los pesos de las n capas ocultas de la red (Optimizacion, evita la redimiension del vector cada vez que se inserta).
	Matrices.reserve(h.size() + 2);
	this->pesos_hn.reserve(h.size() - 1);
	this->gradientes.reserve(h.size() - 1);
	this->deltas.reserve(h.size() - 1);
	this->salidas_capas_ocultas.reserve(h.size());
	this->errores.reserve(h.size());
	this->bias.reserve(h.size() + 1);
	Matrices.push_back(pesos_ih);

	for (int i = 0; i < h.size(); i++)
	{
		errores.push_back(nullptr);
		salidas_capas_ocultas.push_back(nullptr);
		this->bias.push_back(new Matrix(h[i], 1));
		this->bias.at(i)->aleatorizar();
	}
	//reserva espacio en memoria para los sesgos de las n capas ocultas y de salida de la red (Optimizacion, evita la redimiension del vector cada vez que se inserta).

	for (size_t i = 0; i < h.size() - 1; i++)
	{
		this->gradientes.push_back(nullptr);
		this->deltas.push_back(nullptr);
		this->pesos_hn.push_back(new Matrix(h[i + 1], h[i]));
		this->pesos_hn.at(i)->aleatorizar();
		Matrices.push_back(pesos_hn[i]);
	}
	//Matriz que representa los pesos entre las capa enesima(oculta)-Salida y se aleatoriza
	this->pesos_ho = new Matrix(this->outputLayerNodes, h[h.size() - 1]);
	this->pesos_ho->aleatorizar();
	Matrices.push_back(pesos_ho);
	//Se asigna un sesgo o predisposicion a la enesima capa oculta y se aleatoriza.
	//Matriz que representa el sesgo de la capa oculta y se aleatoriza
	this->bias.push_back(new Matrix(outputLayerNodes, 1));
	this->bias.at(bias.size() - 1)->aleatorizar();
	//Variables por eliminar
	this->hiddenLayerNodes = 0;
	this->salidas_capa_oculta = nullptr;
	this->bias_h = nullptr;
	this->bias_o = nullptr;
	prepareCuda();
}

NeuralNetwork::~NeuralNetwork() {
	delete(pesos_ih);
	delete(pesos_ho);
	for (auto& matrix : this->errores) {
		delete matrix;
	}
	for (auto& matrix : this->gradientes) {
		delete matrix;
	}
	for (auto& matrix : this->deltas) {
		delete matrix;
	}
	for (auto& matrix : this->salidas_capas_ocultas) {
		delete matrix;
	}
}

std::vector<float>* NeuralNetwork::feedForwardDNN(std::vector<float>* vec_entrada) {
	//----Generando las salidas de la capas ocultas----
	//Convierte el vector de entrada a una matriz
	Matrix* entradas = Matrix::fromVector(vec_entrada);
	//Se multiplica la matriz de pesos entre la capa oculta y la matriz de entradas
	Matrix* entradas_capa_oculta = Matrix::multiplicar(this->pesos_ih, entradas);
	//Al resultado de la multiplicacion se le agrega el sesgo
	entradas_capa_oculta->suma(this->bias.at(0));
	//sig((W * i) + b) se aplica la funcion sigmoide
	////this->salidas_capas_ocultas.push_back(Matrix::map(entradas_capa_oculta, sigmoid));

	this->salidas_capas_ocultas.at(0) = Matrix::map(entradas_capa_oculta, sigmoid);
	//Se multiplica la matriz de pesos entre la capas ocultas y la matriz de entradas previas
	for (size_t i = 0; i < (hiddenLayerSize - 1); i++)
	{
		//Se multiplica la matriz de pesos entre la capa oculta y la matriz de entradas de la enesima capa oculta
		////this->salidas_capas_ocultas.push_back(Matrix::multiplicar(this->pesos_hn.at(i), this->salidas_capas_ocultas.at(i)));
		this->salidas_capas_ocultas.at(i + 1) = Matrix::multiplicar(this->pesos_hn.at(i), this->salidas_capas_ocultas.at(i));
		//Al resultado de la multiplicacion se le agrega el sesgo
		this->salidas_capas_ocultas.at(i + 1)->suma(this->bias.at(i + 1));
		//sig((W * i) + b) se aplica la funcion sigmoide
		this->salidas_capas_ocultas.at(i + 1)->map(sigmoid);
	}
	//----Generando las salida----
	//Se multiplica la matriz de pesos entre la capa de salida y la matriz de salidas de la capa oculta
	Matrix* entradas_capa_salida = Matrix::multiplicar(this->pesos_ho, this->salidas_capas_ocultas.at(salidas_capas_ocultas.size() - 1));
	//Al resultado de la multiplicacion se le agrega el sesgo
	entradas_capa_salida->suma(bias.at(hiddenLayerSize));
	//sig((W * i) * b) se aplica la funcion sigmoide
	Matrix* salidas = Matrix::map(entradas_capa_salida, sigmoid);
	//delete entradas_capa_oculta;
	return Matrix::toVector(salidas);
}

std::vector<float>* NeuralNetwork::feedForward(std::vector<float>* vec_entrada) {
	/*
		Generando las salidas de la capa oculta
	*/
	//Convierte el vector de entrada a una matriz
	Matrix* entradas = Matrix::fromVector(vec_entrada);
	//Se multiplica la matriz de pesos entre la capa oculta y la matriz de entradas
	Matrix* entradas_capa_oculta = Matrix::multiplicar(this->pesos_ih, entradas);
	//Al resultado de la multiplicacion se le agrega el sesgo
	entradas_capa_oculta->suma(this->bias_h);
	//sig((W * i) + b) se aplica la funcion sigmoide
	this->salidas_capa_oculta = Matrix::map(entradas_capa_oculta, sigmoid);
	/*
		Generando la salida
	*/
	//Se multiplica la matriz de pesos entre la capa de salida y la matriz de salidas de la capa oculta
	Matrix* entradas_capa_salida = Matrix::multiplicar(this->pesos_ho, this->salidas_capa_oculta);
	//Al resultado de la multiplicacion se le agrega el sesgo
	entradas_capa_salida->suma(this->bias_o);
	//sig((W * i) * b) se aplica la funcion sigmoide
	Matrix* salidas = Matrix::map(entradas_capa_salida, sigmoid);
	return Matrix::toVector(salidas);
}

void NeuralNetwork::train(std::vector<float>* vec_entradas, std::vector<float>* vec_respuestas) {
	std::vector<float>* vec_salidas = this->feedForward(vec_entradas);
	//Convertir vectores a matrices
	Matrix* entradas = Matrix::fromVector(vec_entradas);
	Matrix* respuestas = Matrix::fromVector(vec_respuestas);
	Matrix* salidas = Matrix::fromVector(vec_salidas);

	//Calcular el error => respuestas - salidas
	Matrix* errores_salida = Matrix::restaElementWise(respuestas, salidas);

	//Calcular los errores de la capa oculta->salida 
	Matrix* errores_capa_oculta_salida = Matrix::multiplicar(Matrix::transpuesta(pesos_ho), errores_salida);

	//Calcular el gradiente de la capa de salida = learning_rate * errores_salida * dsigmoid(salidas)
	Matrix* gradiente_salida = Matrix::map(salidas, dsigmoid);
	gradiente_salida->productoHadamard(errores_salida);
	gradiente_salida->productoScalar(learning_rate);

	//Calcular los gradientes de la capa oculta = learning_rate * errores_capa_oculta * dsigmoid(salidas_capa_oculta)
	Matrix* gradientes_capa_oculta = Matrix::map(this->salidas_capa_oculta, dsigmoid);
	gradientes_capa_oculta->productoHadamard(errores_capa_oculta_salida);
	gradientes_capa_oculta->productoScalar(learning_rate);

	//Calcular deltas de la capa oculta-salida
	//pesos_delta = learning_rate * errores * dsigmoid(salidas) * pesos(T)
	Matrix* deltas_pesos_ho = Matrix::multiplicar(gradiente_salida, Matrix::transpuesta(this->salidas_capa_oculta));
	this->pesos_ho->suma(deltas_pesos_ho);
	this->bias_o->suma(gradiente_salida);

	//Calcular deltas de la capa de entrada-oculta
	Matrix* deltas_pesos_ih = Matrix::multiplicar(gradientes_capa_oculta, Matrix::transpuesta(entradas));
	this->pesos_ih->suma(deltas_pesos_ih);
	this->bias_h->suma(gradientes_capa_oculta);
}

void NeuralNetwork::trainDNN(std::vector<float>* vec_entradas, std::vector<float>* vec_respuestas) {
	std::vector<float>* vec_salidas = this->feedForwardDNN(vec_entradas);
	//Convertir vectores a matrices
	Matrix* entradas = Matrix::fromVector(vec_entradas);
	Matrix* respuestas = Matrix::fromVector(vec_respuestas);
	Matrix* salidas = Matrix::fromVector(vec_salidas);

	//Calcular el error => respuestas - salidas
	Matrix* errores_salida = Matrix::restaElementWise(respuestas, salidas);

	//Calcular los errores de la capa oculta->salida 

	this->errores.at(hiddenLayerSize - 1) = Matrix::multiplicar(Matrix::transpuesta(this->pesos_ho), errores_salida);
	for (int i = hiddenLayerSize - 1; i > 0; i--)
	{
		this->errores.at(i - 1) = Matrix::multiplicar(Matrix::transpuesta(this->pesos_hn.at(i - 1)), errores.at(i));
	}

	//Calcular el gradiente de la capa de salida = learning_rate * errores_salida * dsigmoid(salidas)
	Matrix* gradiente_salida = Matrix::map(salidas, dsigmoid);
	gradiente_salida->productoHadamard(errores_salida);
	gradiente_salida->productoScalar(learning_rate);
	this->bias.at(bias.size() - 1)->suma(gradiente_salida);
	//Calcular el gradiente de las capas ocultas y oculta-salida = learning_rate * errores_salida * dsigmoid(salidas)
	for (int i = hiddenLayerSize - 1; i > 0; i--)
	{
		this->gradientes.at(i - 1) = Matrix::map(this->salidas_capas_ocultas.at(hiddenLayerSize - i), dsigmoid);
		this->gradientes.at(i - 1)->productoHadamard(errores.at(hiddenLayerSize - i));
		this->gradientes.at(i - 1)->productoScalar(learning_rate);
		this->bias.at(hiddenLayerSize - i)->suma(gradientes.at(i - 1));
	}
	//Calcular los gradientes de la capa entada-oculta = learning_rate * errores_capa_oculta * dsigmoid(salidas_capa_oculta)
	Matrix* gradiente_entrada_oculta = Matrix::map(this->salidas_capas_ocultas.at(0), dsigmoid);
	gradiente_entrada_oculta->productoHadamard(errores.at(0));
	gradiente_entrada_oculta->productoScalar(learning_rate);
	this->bias.at(0)->suma(gradiente_entrada_oculta);

	//Calcular deltas de la capa oculta-salida
	Matrix* deltas_pesos_ho = Matrix::multiplicar(gradiente_salida, Matrix::transpuesta(this->salidas_capas_ocultas.at(salidas_capas_ocultas.size() - 1)));
	this->pesos_ho->suma(deltas_pesos_ho);
	for (int i = hiddenLayerSize - 1; i > 0; i--)
	{
		//this->deltas.at(i-1) = Matrix::multiplicar(gradientes.at(i-1), Matrix::transpuesta(this->salidas_capas_ocultas.at(i)));
		this->deltas.at(i - 1) = Matrix::multiplicar(this->salidas_capas_ocultas.at(i), Matrix::transpuesta(gradientes.at(i - 1)));
		this->pesos_hn.at(i - 1)->suma(deltas.at(i - 1));
	}
	//Calcular deltas de la capa de entrada-oculta
	Matrix* deltas_pesos_ih = Matrix::multiplicar(gradiente_entrada_oculta, Matrix::transpuesta(entradas));
	this->pesos_ih->suma(deltas_pesos_ih);
}

float NeuralNetwork::sigmoid(float n) {
	return (1 / (1 + pow(2.718281828, (-n))));
}

float NeuralNetwork::dsigmoid(float y) {
	//return sigmoid(n) * (1 - sigmoid(n));
	return (y * (1 - y));
}

int main() {
	srand(static_cast<unsigned>(time(0)));
	NeuralNetwork* nn = new NeuralNetwork(2, 4, 1);
	std::vector<float> entradas[] = { { 0 , 0},
									  { 1 , 0},
									  { 0 , 1},
									  { 1 , 1} };
	std::vector<float> esperado[] = { {0},{1},{1},{0} };
	for (size_t i = 0; i < 30000; i++)
	{
		int index = rand() % 4;
		nn->train(&entradas[index], &esperado[index]);
	}
	std::cout << "0,0: " << nn->feedForward(&entradas[0])->at(0) << std::endl;
	std::cout << "0,1: " << nn->feedForward(&entradas[1])->at(0) << std::endl;
	std::cout << "1,0: " << nn->feedForward(&entradas[2])->at(0) << std::endl;
	std::cout << "1,1: " << nn->feedForward(&entradas[3])->at(0) << std::endl;

	std::cout << std::endl;
	std::vector<int> f1 = { 4 };
	NeuralNetwork* nn2 = new NeuralNetwork(2, f1, 1);
	for (size_t i = 0; i < 15000; i++)
	{
		int index = rand() % 4;
		nn2->trainDNN(&entradas[index], &esperado[index]);
	}
	std::cout << "0,0: " << nn2->feedForwardDNN(&entradas[0])->at(0) << std::endl;
	std::cout << "0,1: " << nn2->feedForwardDNN(&entradas[1])->at(0) << std::endl;
	std::cout << "1,0: " << nn2->feedForwardDNN(&entradas[2])->at(0) << std::endl;
	std::cout << "1,1: " << nn2->feedForwardDNN(&entradas[3])->at(0) << std::endl;
	return 0;
}

//hipError_t cudaStatus;
//// Check for any errors launching the kernel
//cudaStatus = hipGetLastError();
//if (cudaStatus != hipSuccess) {
//    fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
//    goto Error;
//}
//
//// hipDeviceSynchronize waits for the kernel to finish, and returns
//// any errors encountered during the launch.
//cudaStatus = hipDeviceSynchronize();
//if (cudaStatus != hipSuccess) {
//    fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
//    goto Error;
//}